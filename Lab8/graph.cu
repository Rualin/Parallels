#include "hip/hip_runtime.h"
#include <iostream>
#include <boost/program_options.hpp>
#include <new>
#include <nvtx3/nvToolsExt.h>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <iomanip>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <cub/block/block_load.cuh>
#include <cub/block/block_reduce.cuh>
#include <cub/block/block_store.cuh>


using vd = double*;
int n, max_iters;
double eps;

#define ind(i, j) ((i) * n + (j))
#define cind(i, j, m) ((i) * (m) + (j))
#define max(a, b) ((a) > (b) ? (a) : (b))
#define abs(a) ((a) < 0 ? (0-(a)) : (a))
#define graph_step 512

// указатель для управления памятью на устройстве
template<typename T>
using cuda_unique_ptr = std::unique_ptr<T, std::function<void(T*)>>;

void f_exception(std::string message) {
    printf("%s!\n", message.c_str());
    exit(2);
}

// выделение памяти на устройстве
template<typename T>
T* cuda_new(size_t size) {
    T *d_ptr;
    hipError_t status;
    status = hipMalloc((void **)&d_ptr, sizeof(T) * size);
    if (status != hipSuccess) f_exception(std::string("hipMalloc error"));
    return d_ptr;
}
// освобождение ресурсов
template<typename T>
void cuda_free(T *dev_ptr) {
    hipError_t status;
    status = hipFree(dev_ptr);
    if (status != hipSuccess) f_exception(std::string("hipFree error"));
}

__global__ void sub_mats(const double *A, const double *Anew, double *subtr_res, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if ((i >= 0) && (i < m) && (j >= 0) && (j < m))
        subtr_res[cind(i, j, m)] = fabs(A[cind(i, j, m)] - Anew[cind(i, j, m)]);
}

__global__ void calc_mean(double *A, double *Anew, int m, bool flag) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (flag) {
        if ((i > 0) && (i < m - 1) && (j > 0) && (j < m - 1))
            A[cind(j, i, m)] = 0.25 * (Anew[cind(j, i + 1, m)] + \
            Anew[cind(j, i - 1, m)] + Anew[cind(j - 1, i, m)] + Anew[cind(j + 1, i, m)]);
    }
    else {
        if ((i > 0) && (i < m - 1) && (j > 0) && (j < m - 1))
            Anew[cind(j, i, m)] = 0.25 * (A[cind(j, i + 1, m)] + \
            A[cind(j, i - 1, m)] + A[cind(j - 1, i, m)] + A[cind(j + 1, i, m)]);
    }
}

void init_grids(std::unique_ptr<double[]> &A, std::unique_ptr<double[]> &Anew) {
    nvtxRangePushA("init");
    memset(A.get(), 0, n * n * sizeof(double));
    //  10 ... 20
    // ... ... ...
    //  20 ... 30
    A[ind(0, 0)] = 10;
    A[ind(0, n - 1)] = 20;
    A[ind(n - 1, n - 1)] = 30;
    A[ind(n - 1, 0)] = 20;
    double dx = 10.0 / (n - 1);
    for (int i = 1; i < n - 1; i++) {
        A[ind(i, 0)] = 10.0 + dx * (double)i;
        A[ind(i, n - 1)] = 20.0 + dx * (double)i;
        A[ind(0, i)] = 10.0 + dx * (double)i;
        A[ind(n - 1, i)] = 20.0 + dx * (double)i;
    }
    std::memcpy(Anew.get(), A.get(), n * n * sizeof(double));
    nvtxRangePop();
}

void create_graph(hipStream_t& stream, hipGraph_t& graph, hipGraphExec_t& instance, double* d_A, double* d_Anew) {
    hipError_t cudaErr = hipSuccess;
    dim3 grid(32, 32);
    dim3 block(32, 32);
    nvtxRangePushA("createGraph");
    // начало захвата операций на потоке stream
    cudaErr = hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    if (cudaErr != hipSuccess) f_exception(std::string("hipStreamBeginCapture error"));
    for (int i = 0; i < graph_step; i++)
        calc_mean<<<grid, block, 0, stream>>>(d_A, d_Anew, n, (i % 2 == 1));
    // завершение захвата операций
    cudaErr = hipStreamEndCapture(stream, &graph);
    if (cudaErr != hipSuccess) f_exception(std::string("hipStreamEndCapture error"));
    nvtxRangePop();
    // создаем исполняемый граф
    cudaErr = hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    if (cudaErr != hipSuccess) f_exception(std::string("hipGraphInstantiate error"));
}

int parse_args(int argc, char** argv) {
    boost::program_options::options_description desc("Heat Equation Solver Options");
    desc.add_options()
        ("help", "help message")
        ("n", boost::program_options::value<int>()->default_value(256), "grid size")
        ("eps", boost::program_options::value<double>()->default_value(1.0e-6), "precision")
        ("iter", boost::program_options::value<int>()->default_value(1000000), "max iterations")
        ("profile", "enable profiling");
    
    boost::program_options::variables_map vm;
    try {
        boost::program_options::store(boost::program_options::parse_command_line(argc, argv, desc), vm);
        boost::program_options::notify(vm);
        
        if (vm.count("help")) {
            std::cout << desc << "\n";
            return 1;
        }
        
        n = vm["n"].as<int>();
        eps = vm["eps"].as<double>();
        max_iters = vm["iter"].as<int>();
        
        if (vm.count("profile")) {
            max_iters = 50;  // for profiling
            std::cout << "PROFILING MODE\n";
        }
    } catch (const std::exception& e) {
        std::cout << "Error: " << e.what() << "\n";
        return 2;
    }
    return 0;
}

int main(int argc, char **argv) {
    switch (parse_args(argc, argv)) {
        case 1:
            return 0;
        case 2:
            return 1;
        default:
            break;
    }

    std::unique_ptr<double[]> A_ptr(new double[n * n]);
    std::unique_ptr<double[]> Anew_ptr(new double[n * n]);

    init_grids(A_ptr, Anew_ptr);
    double* A = A_ptr.get();
    double* Anew = Anew_ptr.get();

    std::string cudaMalloc_err = "hipMalloc error";
    std::string cudaMemcpy_err = "hipMemcpy error";
    std::string cudaGraphLaunch_err = "hipGraphLaunch error";
    std::string cudaStreamCreate_err = "hipStreamCreate error";

    dim3 grid(32, 32);
    dim3 block(32, 32);

    hipError_t cudaErr = hipSuccess;
    hipStream_t stream;
    
    cudaErr = hipStreamCreate(&stream);
    if (cudaErr != hipSuccess) f_exception(cudaStreamCreate_err);  

    cuda_unique_ptr<double> d_unique_ptr_error(cuda_new<double>(1), cuda_free<double>);
    cuda_unique_ptr<void> d_unique_ptr_temp_storage(cuda_new<void>(0), cuda_free<void>);

    cuda_unique_ptr<double> d_unique_ptr_A(cuda_new<double>(n*n), cuda_free<double>);
    cuda_unique_ptr<double> d_unique_ptr_Anew(cuda_new<double>(n*n), cuda_free<double>);
    cuda_unique_ptr<double> d_unique_ptr_subtr_temp(cuda_new<double>(n*n), cuda_free<double>);

    // выделение памяти и перенос на устройство
	double *d_error_ptr = d_unique_ptr_error.get();
    double *d_A = d_unique_ptr_A.get();
	double *d_Anew = d_unique_ptr_Anew.get();
    double *d_subtr_temp = d_unique_ptr_subtr_temp.get();

    // копирование матриц с хоста на gpu
    cudaErr = hipMemcpy(d_A, A, n * n * sizeof(double), hipMemcpyHostToDevice);
    if (cudaErr != hipSuccess) f_exception(cudaMemcpy_err);
    cudaErr = hipMemcpy(d_Anew, Anew, n * n * sizeof(double), hipMemcpyHostToDevice);
    if (cudaErr != hipSuccess) f_exception(cudaMemcpy_err);

    // проверка памяти для редукции
    void *d_temp_storage = d_unique_ptr_temp_storage.get();
    size_t temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_Anew, d_error_ptr, n * n, stream);
    cudaErr = hipMalloc((void**)&d_temp_storage, temp_storage_bytes);
    if (cudaErr != hipSuccess) f_exception(cudaMalloc_err);

    // printf("Jacobi relaxation Calculation: %d x %d mesh\n", n, n);

    hipGraph_t graph;
    hipGraphExec_t instance;

    int iter = 0;
    double error = eps + 1.0;
    create_graph(stream, graph, instance, d_A, d_Anew);

    nvtxRangePushA("while");
    auto start_time = std::chrono::steady_clock::now();
    while (error > eps && iter < max_iters) {
        // старт графа
        nvtxRangePushA("startGraph");
        cudaErr = hipGraphLaunch(instance, stream);
        if (cudaErr != hipSuccess) f_exception(cudaGraphLaunch_err);
        nvtxRangePop();

        iter += graph_step;
        if (iter % graph_step == 0) {
            nvtxRangePushA("calcError");
            sub_mats<<<grid, block, 0, stream>>>(d_A, d_Anew, d_subtr_temp, n);
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_subtr_temp, d_error_ptr, n * n, stream);
            cudaErr = hipMemcpy(&error, d_error_ptr, sizeof(double), hipMemcpyDeviceToHost);
            if (cudaErr != hipSuccess) f_exception(cudaMemcpy_err);
            nvtxRangePop();
        }
    }
    nvtxRangePop();
    auto end_time = std::chrono::steady_clock::now();
    std::chrono::duration<double> dur = end_time - start_time;

    cudaErr = hipMemcpy(A, d_A, n * n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaErr != hipSuccess) f_exception(cudaMemcpy_err);

    // освобождение ресурсов
    hipStreamDestroy(stream);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(instance);

    std::cout << "Iters: " << iter << "\n";
    std::cout << "Error: " << error << "\n";
    std::cout << "Elapsed time: " << dur.count() << "\n";
    return 0;
}